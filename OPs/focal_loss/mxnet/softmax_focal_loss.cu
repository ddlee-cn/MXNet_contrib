#include "hip/hip_runtime.h"
// Modified from Official Caffe2 implementation
// Author: ddlee, me@ddlee.cn

#include "./softmax_focal_loss-inl.h"
#include <mshadow/tensor.h>

namespace mshadow {
namespace cuda {

template<typename DType>
__global__ void SpatialSoftmaxKernel(const int N, const int A,
    const int H, const int W, const float* Xdata, float* Pdata,
    const int num_classes) {
    CUDA_1D_KERNEL_LOOP(index, N * A * H * W) {
    int D = num_classes * A;
    int x = index % W;
    int y = (index / W) % H;
    int a = (index / (W * H)) % A;
    int i = index / W / H / A;

    // Subtract max on each cell for numerical reasons
    float max_val = -FLT_MAX;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
        int idx = i * (H * W * D) +  c * (H * W) + y * W + x;
        max_val = max(max_val, Xdata[idx]);
    }
    // Exponentiate
    float expsum = 0.0f;
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
        int idx = i * (H * W * D) + c * (H * W) + y * W + x;
        float expx = exp(Xdata[idx] - max_val);
        Pdata[idx] = expx;
        expsum += expx;
    }
    // Normalize
    for(int c = a * num_classes; c < (a + 1) * num_classes; ++c) {
        int idx = i * (H * W * D) + c * (H * W) + y * W + x;
        Pdata[idx] /= expsum;
    }
    }
}

template<typename DType>
__global__ void SoftmaxFocalLossKernel(
    const int N, const int A, const int H, const int W,
    const float* Pdata, const int* targets, float* losses,
    const float* weight_pos, const float gamma, const float alpha,
    const int num_classes) {
    CUDA_1D_KERNEL_LOOP(i, N * A * H * W) {
    int D = A * num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int a = (i / (W * H)) % A;
    int n = i / (W * H * A);
    const int label = static_cast<int>(targets[i]);

    float Np = max(weight_pos[0], 1.0);
    float z = (label == 0) * (1 - alpha) / Np +
                (label >= 1) * alpha / Np;

    losses[i] = 0.0;
    if (label >= 0) {
        int offset = a * num_classes;
        int idx = n * (H * W * D) + (offset + label) * (H * W) + y * W + x;
        losses[i] =
            -(pow(1.0 - Pdata[idx], gamma) *
            log(max(Pdata[idx], FLT_MIN))) * z;
    }
    }
}

template<typename DType>
__global__ void SoftmaxFocalLossGradientWeightKernel(
    const int N, const int A, const int H, const int W,
    const float* Pdata, const int* targets, float* buff,
    const float* weight_pos, const float gamma, const float alpha,
    const int num_classes) {
    CUDA_1D_KERNEL_LOOP(i, N * A * H * W) {
    int D = A * num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int a = (i / (W * H)) % A;
    int n = i / (W * H * A);
    const int label = static_cast<int>(targets[i]);
    float Np = max(weight_pos[0], 1.0);
    float z =  (label == 0) * (1 - alpha) / Np +
                (label >= 1) * alpha / Np;

    buff[i] = 0.0;
    if (label >= 0) {
        int offset = a * num_classes;
        int idx = n * (H * W * D) + (offset + label) * (H * W) + y * W + x;
        float onemp = 1. - Pdata[idx];
        float p = Pdata[idx];
        buff[i] =
            (-pow(onemp, gamma) +
            gamma * pow(onemp, gamma - 1) * p * log(max(p, FLT_MIN))) * z;
    }
    }
}

template<typename DType>
__global__ void SoftmaxFocalLossGradientKernel(
    const int N, const int D, const int H, const int W,
    const float* Pdata, const int* targets, const float* buff,
    const float* d_loss_data, float* dX, const int num_classes) {
    CUDA_1D_KERNEL_LOOP(i, N * D * H * W) {
    int A = D / num_classes;
    int x = i % W;
    int y = (i / W) % H;
    int d = (i / (W * H)) % D;
    int a = d / num_classes;
    int c = d % num_classes;
    int n = i / (W * H * D);
    float d_loss = *d_loss_data;

    int ind = n * (H * W * A) + a * (H * W) + y * W + x;
    const int label = static_cast<int>(targets[ind]);

    float c1 = (label >= 0) * 1.0;
    float c2 = (label == c) * 1.0;
    dX[i] = 0.0;
    dX[i] = c1 * d_loss * buff[ind] * (c2 - Pdata[i]);
    }
}
} // cuda


template<typename Dtype>
inline void SoftmaxFocalLossForward(const Tensor<gpu, 4, Dtype> &X, // Logits
                            const Tensor<gpu, 4, Dtype> &T, // Labels

) {
  auto& X = Input(0);         // Logits
  auto& T = Input(1);         // Labels
  auto& wp = Input(2);        // num of foregound
  auto* avg_loss = Output(0); // average loss as output
  auto* P = Output(1);        // softmax probability, going to be re-used in gradient

  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);
  int A = D / num_classes_;

  losses_.Resize(N * A * H * W);
  P->Resize(N * D * H * W);
  avg_loss->Resize(vector<TIndex>());
  math::Set<float, HIPContext>(
      avg_loss->size(), 0.f, avg_loss->mutable_data<float>(), &context_);
  math::Set<float, HIPContext>(
      P->size(), 0.f, P->mutable_data<float>(), &context_);
  math::Set<float, HIPContext>(
      losses_.size(), 0.f, losses_.mutable_data<float>(), &context_);
  DCHECK_EQ(X.ndim(), 4);

  const float* Xdata = X.data<float>();
  const float* Wdata = wp.data<float>();

// Labels
  // Spatial Softmax Kernel
  SpatialSoftmaxKernel
      <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, A, H, W, Xdata, P->mutable_data<float>(), num_classes_);

  // Compute loss for each x,y location
  const int* Tdata = T.data<int>();
  SoftmaxFocalLossKernel
  <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
      0, context_.hip_stream()>>>(
    N, A, H, W, P->data<float>(), Tdata, losses_.mutable_data<float>(),
    Wdata, gamma_, alpha_, num_classes_);

  // sum the losses
  float* avg_loss_data = avg_loss->mutable_data<float>();
  math::Sum<float, HIPContext>(
      losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
  math::Scale<float, HIPContext>(
      1, scale_, avg_loss_data, avg_loss_data, &context_);

  return true;
}


template<>
bool SoftmaxFocalLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);    // Logits
  auto& T = Input(1);    // Label
  auto& wp = Input(2);   // num of foreground example
  auto& P = Input(3);    // Softmax Probability
  auto& d_avg_loss = Input(4);
  auto* dX = Output(0);  // gradient wrt logits


  int N = X.dim32(0);
  int D = X.dim32(1);
  int H = X.dim32(2);
  int W = X.dim32(3);
  int A = D / num_classes_;

  buff_.Resize(N * A * H * W);

  dX->ResizeLike(X);

  const float* Xdata = X.data<float>();
  const int* Tdata = T.data<int>();
  const float* Pdata = P.data<float>();
  const float* Wdata = wp.data<float>();


  // Compute the weight for gradients
  SoftmaxFocalLossGradientWeightKernel
      <<<CAFFE_GET_BLOCKS(N * A * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, A, H, W, Pdata, Tdata, buff_.mutable_data<float>(),
    Wdata, gamma_, alpha_, num_classes_);
  // Compute the gradient with the weights
  const float* Bdata = buff_.data<float>();
  SoftmaxFocalLossGradientKernel
      <<<CAFFE_GET_BLOCKS(N * D * H * W), CAFFE_CUDA_NUM_THREADS,
         0, context_.hip_stream()>>>(
    N, D, H, W, Pdata, Tdata, Bdata, d_avg_loss.data<float>(),
    dX->mutable_data<float>(), num_classes_);
  math::Scale<float, HIPContext>(
    dX->size(), scale_, dX->data<float>(), dX->mutable_data<float>(),
    &context_);
  return true;
}

} // mshadow



namespace mxnet {
namespace op {

template<>
Operator* CreateOp<gpu>(SoftmaxFocalLossParam param, int dtype) {
    Operator* op = NULL;
    MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new SoftmaxFocalLossOp<gpu, DType>(param);
    });
    return op;
}

}  // namespace op
}  // namespace mxnet