#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2018 University of Maryland, College Park
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file multi_proposal_target.cc
 * \brief Proposal target layer
 * \author Bharat Singh
*/

#include "./multi_proposal_target-inl.h"
#include <set>
#include <math.h>
#include <unistd.h>
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "./operator_common.h"
#include "./mshadow_op.h"
#include <time.h>
#include <stdlib.h> 
//============================
// Bounding Box Transform Utils
//============================

#define NUM_THREADS_NMS 1024

namespace mxnet {
namespace op {
namespace utils {



// filter box by set confidence to zero
// * height or width < rpn_min_size
inline void FilterBox(float *dets,
                      int num_dets, float min_size) {
  #pragma omp parallel for num_threads(8)
  for (int i = 0; i < num_dets; ++i) {
    float iw = dets[5*i + 2] - dets[5*i] + 1.0f;
    float ih = dets[5*i + 3] - dets[5*i + 1] + 1.0f;
    if (iw < min_size || ih < min_size) {
      dets[5*i+0] -= min_size / 2;
      dets[5*i+1] -= min_size / 2;
      dets[5*i+2] += min_size / 2;
      dets[5*i+3] += min_size / 2;
      dets[5*i+4] = -1.0f;
    }
  }
}


inline void _MakeAnchor(float w,
                        float h,
                        float x_ctr,
                        float y_ctr,
                        std::vector<float> *out_anchors) {
  out_anchors->push_back(x_ctr - 0.5f * (w - 1.0f));
  out_anchors->push_back(y_ctr - 0.5f * (h - 1.0f));
  out_anchors->push_back(x_ctr + 0.5f * (w - 1.0f));
  out_anchors->push_back(y_ctr + 0.5f * (h - 1.0f));
}

inline void _Transform(float scale,
                       float ratio,
                       const std::vector<float>& base_anchor,
                       std::vector<float>  *out_anchors) {
  float w = base_anchor[2] - base_anchor[0] + 1.0f;
  float h = base_anchor[3] - base_anchor[1] + 1.0f;
  float x_ctr = base_anchor[0] + 0.5 * (w - 1.0f);
  float y_ctr = base_anchor[1] + 0.5 * (h - 1.0f);
  float size = w * h;
  float size_ratios = std::floor(size / ratio);
  float new_w = std::floor(std::sqrt(size_ratios) + 0.5f) * scale;
  float new_h = std::floor((new_w / scale * ratio) + 0.5f) * scale;

  _MakeAnchor(new_w, new_h, x_ctr,
             y_ctr, out_anchors);
}

// out_anchors must have shape (n, 5), where n is ratios.size() * scales.size()
inline void GenerateAnchors(const std::vector<float>& base_anchor,
                            const nnvm::Tuple<float>& ratios,
                            const nnvm::Tuple<float>& scales,
                            std::vector<float> *out_anchors) {

  for (size_t j = 0; j < ratios.ndim(); ++j) {
    for (size_t k = 0; k < scales.ndim(); ++k) {
      _Transform(scales[k], ratios[j], base_anchor, out_anchors);
    }
  }
}

// greedily keep the max detections
__global__ void NonMaximumSuppression(float* dets,
                                  int post_nms_top_n,
                                  int num_images,
                                  int num_anchors,
                                  int width,
                                  int height,
                                  float* propsout) {
  
  int i = blockIdx.x;
  int t = threadIdx.x;
  int chip_anchors = num_anchors*width*height;
  int num_threads = blockDim.x;
  int chip_index = i*chip_anchors;

  int vct = 0;

  __shared__ float maxbuf[NUM_THREADS_NMS];
  __shared__ int maxidbuf[NUM_THREADS_NMS];
  __shared__ float maxvbuf[32];
  __shared__ int maxidvbuf[32];
  __shared__ float boxbuf[6];

  for (int j = chip_index; j < chip_index + chip_anchors && vct < post_nms_top_n; j++) {
    //find max
    float vmax = -2;
    int maxid = j + t;
    for (int k = j + t; k < chip_index + chip_anchors; k = k + num_threads) {
      if (dets[6*k + 4] > vmax) {
        vmax = dets[6*k + 4];
        maxid = k;
      }
    }
    maxbuf[t] = vmax;
    maxidbuf[t] = maxid;
    __syncthreads();

    if (t < 32) {
      float vmax = maxbuf[0];
      int maxid = maxidbuf[0];
      for (int k = t; k < NUM_THREADS_NMS; k = k + 32) {
        if (maxbuf[k] > vmax) {
          vmax = maxbuf[k];
          maxid = maxidbuf[k];
        }
      }
      maxvbuf[t] = vmax;
      maxidvbuf[t] = maxid;
    }
    __syncthreads();

    int basep = chip_index + vct;

    if (t == 0) {
      vmax = maxvbuf[0];
      maxid = maxidvbuf[0];
      for (int k = 0; k < 32; k++) {
        if (maxvbuf[k] > vmax) {
          vmax = maxvbuf[k];
          maxid = maxidvbuf[k];
        }
      }
      //swap it with the kth element
      float tmpx1, tmpx2, tmpy1, tmpy2, tmps, tmpa;
      
      tmpx1 = dets[6*basep];
      tmpy1 = dets[6*basep+1];
      tmpx2 = dets[6*basep+2];
      tmpy2 = dets[6*basep+3];
      tmps = dets[6*basep+4];
      tmpa = dets[6*basep+5];

      dets[6*basep] = dets[6*maxid];
      dets[6*basep+1] = dets[6*maxid+1];
      dets[6*basep+2] = dets[6*maxid+2];
      dets[6*basep+3] = dets[6*maxid+3];
      dets[6*basep+4] = dets[6*maxid+4];
      dets[6*basep+5] = dets[6*maxid+5];

      dets[6*maxid] = tmpx1;
      dets[6*maxid+1] = tmpy1;
      dets[6*maxid+2] = tmpx2;
      dets[6*maxid+3] = tmpy2;
      dets[6*maxid+4] = tmps;
      dets[6*maxid+5] = tmpa;

      boxbuf[0] = dets[6*basep];
      boxbuf[1] = dets[6*basep+1];
      boxbuf[2] = dets[6*basep+2];
      boxbuf[3] = dets[6*basep+3];
      boxbuf[4] = dets[6*basep+4];
      boxbuf[5] = dets[6*basep+5];
    }
    __syncthreads();
      
    //invalidate all boxes with overlap > 0.7 with max box

    float ix1 = boxbuf[0];
    float iy1 = boxbuf[1];
    float ix2 = boxbuf[2];
    float iy2 = boxbuf[3];
    float iarea = boxbuf[5];

    if (boxbuf[4] == -1) {
      break;
    }

    vct = vct + 1;
    float xx1, xx2, yy1, yy2, w, h, inter, ovr;
    for (int pind = j + 1 + t; pind < chip_index + chip_anchors; pind = pind + num_threads) {
      if (dets[6*pind + 4] == -1) {
        continue;
      } 
      xx1 = fmaxf(ix1, dets[6*pind]);
      yy1 = fmaxf(iy1, dets[6*pind + 1]);
      xx2 = fminf(ix2, dets[6*pind + 2]);
      yy2 = fminf(iy2, dets[6*pind + 3]);
      w = fmaxf(0.0f, xx2 - xx1 + 1.0f);
      h = fmaxf(0.0f, yy2 - yy1 + 1.0f);
      inter = w * h;
      ovr = inter / (iarea + dets[6*pind+5] - inter);
      if (ovr > 0.7) {
        dets[6*pind + 4] = -1;
      }
    }
    __syncthreads();
  }

  for (int k = chip_index + vct + t; k < chip_index + post_nms_top_n; k = k + num_threads) {
    dets[6*k] = k % 100;
    dets[6*k + 1] = k% 100;
    dets[6*k + 2] = k % 100 + 200;
    dets[6*k + 3] = k % 100 + 200;
  }
  __syncthreads();

  if (t < post_nms_top_n) {
    propsout[5*(i*post_nms_top_n + t)] = i;
    propsout[5*(i*post_nms_top_n + t) + 1] = dets[6*(chip_index + t)];
    propsout[5*(i*post_nms_top_n + t) + 2] = dets[6*(chip_index + t)+1];
    propsout[5*(i*post_nms_top_n + t) + 3] = dets[6*(chip_index + t)+2];
    propsout[5*(i*post_nms_top_n + t) + 4] = dets[6*(chip_index + t)+3];
  }
  __syncthreads();
}


__global__ void getProps(float* boxes,
                             float* deltas,
                             float* im_info,
                             float* anchorbuf,
                             float* scores,
                             float* valid_ranges,
                             int num_images,
                             int anchors,
                             int heights,
                             int widths,
                             int stride) {
  int num_anchors = anchors * heights * widths;
  int t = blockDim.x * blockIdx.x + threadIdx.x;

  if (t < num_images * num_anchors) {
    
    int b = t / num_anchors;
    int index = t % num_anchors;
    int a = index / (heights*widths);
    int mat = index % (heights*widths);
    int w = mat % widths; //width index
    int h = mat / widths; //height index
    boxes[6*t] = anchorbuf[4*a] + w * stride;
    boxes[6*t + 1] = anchorbuf[4*a+1] + h * stride;
    boxes[6*t + 2] = anchorbuf[4*a+2] + w * stride;
    boxes[6*t + 3] = anchorbuf[4*a+3] + h * stride;
    boxes[6*t + 4] = scores[b*num_anchors*2 + ((anchors + a)*heights + h)*widths + w];

    float width = boxes[6*t + 2] - boxes[6*t] + 1.0;
    float height = boxes[6*t + 3] - boxes[6*t + 1] + 1.0;
    float ctr_x = boxes[6*t + 0] + 0.5 * (width - 1.0);
    float ctr_y = boxes[6*t + 1] + 0.5 * (height - 1.0);
    float dx = deltas[b*num_anchors*4 + a*4*widths*heights + h*widths + w];
    float dy = deltas[b*num_anchors*4 + (a*4 + 1)*widths*heights + h*widths + w];
    float dw = deltas[b*num_anchors*4 + (a*4 + 2)*widths*heights + h*widths + w];
    float dh = deltas[b*num_anchors*4 + (a*4 + 3)*widths*heights + h*widths + w];
    float pred_ctr_x = dx * width + ctr_x;
    float pred_ctr_y = dy * height + ctr_y;
    float pred_w = exp(dw) * width;
    float pred_h = exp(dh) * height;
    float pred_x1 = pred_ctr_x - 0.5 * (pred_w - 1.0);
    float pred_y1 = pred_ctr_y - 0.5 * (pred_h - 1.0);
    float pred_x2 = pred_ctr_x + 0.5 * (pred_w - 1.0);
    float pred_y2 = pred_ctr_y + 0.5 * (pred_h - 1.0);

    pred_x1 = fmaxf(fminf(pred_x1, im_info[3*b+1] - 1.0f), 0.0f);
    pred_y1 = fmaxf(fminf(pred_y1, im_info[3*b] - 1.0f), 0.0f);
    pred_x2 = fmaxf(fminf(pred_x2, im_info[3*b+1] - 1.0f), 0.0f);
    pred_y2 = fmaxf(fminf(pred_y2, im_info[3*b] - 1.0f), 0.0f);
    boxes[6*t] = pred_x1;
    boxes[6*t + 1] = pred_y1;
    boxes[6*t + 2] = pred_x2;
    boxes[6*t + 3] = pred_y2;
    
    int min_size = 3;
    if ((pred_y2 - pred_y1) < min_size && (pred_x2 - pred_x1) < min_size) {
      boxes[6*t] -= min_size/2;
      boxes[6*t + 1] -= min_size/2;
      boxes[6*t + 2] += min_size/2;
      boxes[6*t + 3] += min_size/2;
      boxes[6*t + 4] = -1;
    }
    float area = (boxes[6*t + 2] - boxes[6*t]) * (boxes[6*t + 3] - boxes[6*t + 1]);
    if (area >= valid_ranges[2*b+1] * valid_ranges[2*b+1] || area < valid_ranges[2*b]*valid_ranges[2*b]) {
      boxes[6*t + 4] = -1;  
    }
    boxes[6*t + 5] = area;
  }
}

}  // namespace utils


template<typename xpu>
class MultiProposalTargetGPUOp : public Operator{
 public:
  float *proposals;
  float *im_info;
  float *gt_boxes;
  float *rois;
  float *labels;
  float *bbox_targets;
  float *bbox_weights;
  float *valid_ranges;

  explicit MultiProposalTargetGPUOp(MultiProposalTargetParam param) {
    this->param_ = param;
    int batch_size = 16;//param.batch_size;
    this->proposals = new float[batch_size*21*6*32*32];
    this->im_info = new float[batch_size*3];
    this->gt_boxes = new float[batch_size*100*5];
    this->valid_ranges = new float[batch_size*2];
    this->rois = new float[300*batch_size*5];
    this->labels = new float[300*batch_size];
    this->bbox_targets = new float[300*batch_size*4];
    this->bbox_weights = new float[300*batch_size*4];
    this->param_.workspace = (param_.workspace << 20) / sizeof(float);
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    CHECK_EQ(in_data.size(), 5);
    CHECK_EQ(out_data.size(), 4);
    
    using namespace mshadow;
    using namespace mshadow::expr;
    //clock_t t;
    //t = clock();
    Stream<gpu> *s = ctx.get_stream<gpu>();

    Tensor<gpu, 4> tscores = in_data[proposal::kClsProb].get<gpu, 4, real_t>(s);
    Tensor<gpu, 4> tbbox_deltas = in_data[proposal::kBBoxPred].get<gpu, 4, real_t>(s);
    Tensor<gpu, 2> tim_info = in_data[proposal::kImInfo].get<gpu, 2, real_t>(s);
    Tensor<gpu, 3> tgt_boxes = in_data[proposal::kGTBoxes].get<gpu, 3, real_t>(s);
    Tensor<gpu, 2> tvalid_ranges = in_data[proposal::kValidRanges].get<gpu, 2, real_t>(s);

    int rpn_post_nms_top_n = param_.rpn_post_nms_top_n;
    int num_images = tbbox_deltas.size(0);
    int num_anchors = tbbox_deltas.size(1) / 4;
    int height = tbbox_deltas.size(2);
    int width = tbbox_deltas.size(3);
    int count_anchors = num_anchors*height*width;
    int total_anchors = count_anchors * num_images;

    int bufsize = (total_anchors*6 + num_images*rpn_post_nms_top_n*5 + num_anchors*4)*sizeof(float);
    Tensor<gpu, 1> workspace = ctx.requested[proposal::kTempSpace].get_space_typed<gpu, 1, float>(Shape1(bufsize), s);

    hipMemcpy(im_info, tim_info.dptr_, 3 * sizeof(float) * num_images, hipMemcpyDeviceToHost);
    hipMemcpy(gt_boxes, tgt_boxes.dptr_, 5 * sizeof(float) * num_images * 100, hipMemcpyDeviceToHost);
    hipMemcpy(valid_ranges, tvalid_ranges.dptr_, 2 * sizeof(float) * num_images, hipMemcpyDeviceToHost);

    float* propbuf = workspace.dptr_;
    float* propsout = workspace.dptr_ + total_anchors*6;    
    float* anchorbuf = workspace.dptr_ + total_anchors*6 + num_images*rpn_post_nms_top_n*5;

    std::vector<float> base_anchor(4);
    //usleep(20000000);
    base_anchor[0] = 0.0;
    base_anchor[1] = 0.0;
    base_anchor[2] = param_.feature_stride - 1.0;
    base_anchor[3] = param_.feature_stride - 1.0;

    std::vector<float> anchors;
    utils::GenerateAnchors(base_anchor,
                           param_.ratios,
                           param_.scales,
                           &anchors);
    unsigned int size = num_anchors*4*sizeof(float);
    hipMemcpy(anchorbuf, &anchors[0], size, hipMemcpyHostToDevice);

    //call cuda kernel
    int threadsPerBlock = NUM_THREADS_NMS; 
    int numblocks = (total_anchors/threadsPerBlock) + 1;
    utils::getProps<<<numblocks, threadsPerBlock>>>(propbuf, tbbox_deltas.dptr_, tim_info.dptr_, anchorbuf, tscores.dptr_,
                                                    tvalid_ranges.dptr_, num_images, num_anchors, height, width, param_.feature_stride);
    hipDeviceSynchronize();
    
    utils::NonMaximumSuppression<<<num_images, threadsPerBlock>>>(propbuf, rpn_post_nms_top_n, num_images, num_anchors, width, height, propsout);
    hipDeviceSynchronize();
    hipError_t error;
    error = hipGetLastError();
    if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
    hipMemcpy(rois, propsout, 5*rpn_post_nms_top_n*num_images*sizeof(float), hipMemcpyDeviceToHost);
    
    std::vector <int> numgts_per_image(num_images);
    std::vector <int> sumgts_per_image(num_images);

    for (int i = 0; i < num_images; i++) {
      numgts_per_image[i] = 0;
      for (int j = 0; j < 100; j++) {
        if (gt_boxes[i*100*5 + j*5 + 4] != -1) {
          numgts_per_image[i]++;
        }
      }
      if (i == 0) {
        sumgts_per_image[i] = numgts_per_image[i];
      } else {
        sumgts_per_image[i] = numgts_per_image[i] + sumgts_per_image[i-1];
      }
    }

    #pragma omp parallel for num_threads(8)
    for (int i = 0; i < num_images; i++) {
      for (int j = 0; j < rpn_post_nms_top_n; j++) {
        int basepos = rpn_post_nms_top_n*i + j;
        labels[basepos] = 0;
        bbox_targets[4*basepos] = 1.0;
        bbox_targets[4*basepos + 1] = 1.0;
        bbox_targets[4*basepos + 2] = 1.0;
        bbox_targets[4*basepos + 3] = 1.0;

        bbox_weights[4*basepos] = 0.0;
        bbox_weights[4*basepos + 1] = 0.0;
        bbox_weights[4*basepos + 2] = 0.0;
        bbox_weights[4*basepos + 3] = 0.0;
      }
      int props_this_batch = rpn_post_nms_top_n;

      for (int k = props_this_batch - numgts_per_image[i], j = 0; k < props_this_batch; j++, k++) {
          float w = gt_boxes[i*100*5 + j*5 + 2] - gt_boxes[i*500 + j*5];
          float h = gt_boxes[i*500 + j*5 + 3] - gt_boxes[i*500 + j*5 + 1];
          float area = w*h;
          if (area >= valid_ranges[2*i]*valid_ranges[2*i] && area <= valid_ranges[2*i+1]*valid_ranges[2*i+1]) {
            rois[i*rpn_post_nms_top_n*5 + k*5 + 1] = gt_boxes[i*500 + j*5];
            rois[i*rpn_post_nms_top_n*5 + k*5 + 2] = gt_boxes[i*500 + j*5 + 1];
            rois[i*rpn_post_nms_top_n*5 + k*5 + 3] = gt_boxes[i*500 + j*5 + 2];
            rois[i*rpn_post_nms_top_n*5 + k*5 + 4] = gt_boxes[i*500 + j*5 + 3];
          }
        }
    }
    #pragma omp parallel for num_threads(8)
    for (int imid = 0; imid < num_images; imid++) {
      int tpct = 0;
      int num_gts_this_image = numgts_per_image[imid];
      //std::cout << "gtc " << num_gts_this_image << std::endl;
      int props_this_batch = rpn_post_nms_top_n;
      if (num_gts_this_image > 0) {
        float *overlaps = new float[props_this_batch * num_gts_this_image];
        float *max_overlaps = new float[props_this_batch];
        for (int i = 0; i < props_this_batch; i++) {
          max_overlaps[i] = 0;
        }
        float *max_overlap_ids = new float[props_this_batch];
        std::set <int> positive_label_ids;
        for (int i = 0; i < props_this_batch; i++) {
          max_overlap_ids[i] = 0;
        }

        for (int i = props_this_batch; i < rpn_post_nms_top_n; i++) {
          labels[imid*rpn_post_nms_top_n + i] = -1;
        }
        //get overlaps, maximum overlaps and gt labels
        for (int i = 0; i < numgts_per_image[imid]; i++) {
          float x1 = gt_boxes[imid*500 + i*5];
          float y1 = gt_boxes[imid*500 + i*5 + 1];
          float x2 = gt_boxes[imid*500 + i*5 + 2];
          float y2 = gt_boxes[imid*500 + i*5 + 3];
          int pbase;
          float a1 = (x2 - x1) * (y2 - y1);
          float xx1, yy1, xx2, yy2, w, h, inter, ovr, a2;
          for (int j = 0; j < props_this_batch; j++) {
            pbase = rpn_post_nms_top_n*imid + j;
            xx1 = std::max(x1, rois[pbase*5 + 1]);
            yy1 = std::max(y1, rois[pbase*5 + 2]);
            xx2 = std::min(x2, rois[pbase*5 + 3]);
            yy2 = std::min(y2, rois[pbase*5 + 4]);
            w = std::max(0.0f, xx2 - xx1 + 1.0f);
            h = std::max(0.0f, yy2 - yy1 + 1.0f);
            a2 = (rois[pbase*5 + 3] - rois[pbase*5 + 1]) * (rois[pbase*5 + 4] - rois[pbase*5 + 2]);
            inter = w * h;
            ovr = inter / (a1 + a2 - inter);
            overlaps[i*num_gts_this_image + j] = ovr;

            if (overlaps[i*num_gts_this_image + j] > max_overlaps[j] && overlaps[i*num_gts_this_image + j] > 0.5) {
              max_overlaps[j] = overlaps[i*num_gts_this_image + j];
              max_overlap_ids[j] = i;
              //set labels for positive proposals
              labels[imid*rpn_post_nms_top_n + j] = gt_boxes[imid*500 + i*5 + 4];
              positive_label_ids.insert(j);
              tpct = tpct + 1;
            }
          }
        }
        //p is for proposal and g is for gt, cx is x center and w,h is width and height
        int pid, gtid;
        float gx1, gx2, gy1, gy2, px1, px2, py1, py2;
        float gcx, gcy, gw, gh, pcx, pcy, pw, ph;
        //generate bbox targets for the positive labels
        for (auto it = positive_label_ids.begin(); it !=positive_label_ids.end(); it++) {
          pid = *it;
          int baseid = (imid*rpn_post_nms_top_n + pid);
          bbox_weights[baseid*4] = 1;
          bbox_weights[baseid*4+1] = 1;
          bbox_weights[baseid*4+2] = 1;
          bbox_weights[baseid*4+3] = 1;

          gtid = max_overlap_ids[pid];

          gx1 = gt_boxes[imid*500 + gtid*5];
          gy1 = gt_boxes[imid*500 + gtid*5 + 1];
          gx2 = gt_boxes[imid*500 + gtid*5 + 2];
          gy2 = gt_boxes[imid*500 + gtid*5 + 3];

          gw = gx2 - gx1 + 1;
          gh = gy2 - gy1 + 1;
          gcx = gx1 + gw*0.5;
          gcy = gy1 + gh*0.5;

          px1 = rois[baseid*5 + 1];
          py1 = rois[baseid*5 + 2];
          px2 = rois[baseid*5 + 3];
          py2 = rois[baseid*5 + 4];

          pw = px2 - px1 + 1;
          ph = py2 - py1 + 1;
          pcx = px1 + (pw-1)*0.5;
          pcy = py1 + (ph-1)*0.5;

          bbox_targets[4*baseid] =  10 * (gcx - pcx) / (pw + 1e-7);
          bbox_targets[4*baseid+1] =  10 * (gcy - pcy) / (ph + 1e-7);
          bbox_targets[4*baseid+2] =  5 * log(gw/(pw + 1e-7));
          bbox_targets[4*baseid+3] =  5 * log(gh/(ph + 1e-7));
        }
        delete [] max_overlap_ids;
        delete [] overlaps;
        delete [] max_overlaps;
      }      
    }
    
    Stream<gpu> *so = ctx.get_stream<gpu>();    
    Tensor<gpu, 2> orois = out_data[proposal::kRoIs].get<gpu, 2, real_t>(so);
    Tensor<gpu, 2> olabels = out_data[proposal::kLabels].get<gpu, 2, real_t>(so);
    Tensor<gpu, 2> obbox_targets = out_data[proposal::kBboxTarget].get<gpu, 2, real_t>(so);
    Tensor<gpu, 2> obbox_weights = out_data[proposal::kBboxWeight].get<gpu, 2, real_t>(so);
    hipMemcpy(orois.dptr_, rois, 5*sizeof(float) * num_images*300, hipMemcpyHostToDevice);
    hipMemcpy(olabels.dptr_, labels, sizeof(float) * num_images*300, hipMemcpyHostToDevice);
    hipMemcpy(obbox_targets.dptr_, bbox_targets, 4*sizeof(float) * num_images*300, hipMemcpyHostToDevice);
    hipMemcpy(obbox_weights.dptr_, bbox_weights, 4*sizeof(float) * num_images*300, hipMemcpyHostToDevice);    
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 5);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 4> gscores = in_grad[proposal::kClsProb].get<xpu, 4, real_t>(s);
    Tensor<xpu, 4> gbbox = in_grad[proposal::kBBoxPred].get<xpu, 4, real_t>(s);
    Tensor<xpu, 2> ginfo = in_grad[proposal::kImInfo].get<xpu, 2, real_t>(s);
    Tensor<xpu, 3> ggt_boxes = in_grad[proposal::kGTBoxes].get<xpu, 3, real_t>(s);
    Tensor<xpu, 2> gvalid_ranges = in_grad[proposal::kValidRanges].get<xpu, 2, real_t>(s);

    // can not assume the grad would be zero
    Assign(gscores, req[proposal::kClsProb], 0);
    Assign(gbbox, req[proposal::kBBoxPred], 0);
    Assign(ginfo, req[proposal::kImInfo], 0);
    Assign(ggt_boxes, req[proposal::kGTBoxes], 0);
    Assign(gvalid_ranges, req[proposal::kValidRanges], 0);
  }

 private:
  MultiProposalTargetParam param_;
};  // class MultiProposalOp

template<>
Operator *CreateOp<gpu>(MultiProposalTargetParam param) {
  return new MultiProposalTargetGPUOp<gpu>(param);
}


}  // namespace op
}  // namespace mxnet
